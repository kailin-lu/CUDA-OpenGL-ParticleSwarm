#include "hip/hip_runtime.h"
#include <iostream>
#include "kernel.h"


// Error handing 
void getCUDAError(char const*msg) {
    if (hipGetLastError() != hipSuccess) {
        std::cout << "CUDA ERROR:: " << msg << " " << hipGetErrorString(hipGetLastError()) << std::endl;
    }
}


// Calculate positions of particles and map to buffer 
void calcCUDA(hipGraphicsResource *VBOparticles_CUDA, int NParticles) {
    float *positions;
    hipGraphicsMapResources(1, &VBOparticles_CUDA, 0); 
    size_t num_bytes;

    // Map buffer to write from CUDA  
    hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, VBOparticles_CUDA);  
    
    // Initialize random state  
    hiprandState *state; 
    hipMalloc(&state, sizeof(hiprandState)); 
    init_kernel<<<1,1>>>(state, clock()); 

    // Execute kernel
    // dim3 blockSize(16,16,1); 
    // dim3 gridSize; 
    int blockSize = 32; 
    int gridSize = (NParticles + blockSize - 1) / blockSize; 
    createVertices<<<gridSize, blockSize>>>(positions, state, NParticles); 

    // Unmap buffer 
    hipGraphicsUnmapResources(1, &VBOparticles_CUDA, 0); 
}


// Initialize state for random numbers 
__global__ void init_kernel(hiprandState *state, long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}


// Random positions
__global__ void createVertices(float *positions, hiprandState *state, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        positions[3 * idx + 0] = hiprand_uniform(state); 
        positions[3 * idx + 1] = hiprand_uniform(state); 
        positions[3 * idx + 2] = hiprand_uniform(state); 
    }
}

