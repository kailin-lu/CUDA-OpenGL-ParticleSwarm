#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void initPositions(float *positions, hiprandState *state, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        positions[idx + 0] = hiprand_uniform(state); 
        positions[idx + 1] = hiprand_uniform(state); 
        positions[idx + 2] = hiprand_uniform(state); 
    }
}

// Initialize state for random numbers 
__global__ void init_kernel(hiprandState *state, long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}


void Kernel::kernelInit(float *positions, int N) {
    // Random seed for gpu 
    hiprandState *state; 
    hipMalloc(&state, sizeof(hiprandState)); 
    init_kernel<<<1,1>>>(state, clock()); 

    int blockSize = 32; 
    int gridSize = (blockSize + N - 1) / N; 

    initPositions<<<gridSize, blockSize>>>(positions, state, N); 
}